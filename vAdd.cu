#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void vAdd(int* A, int* B, int* C, int num_elements){

	//Posicion del thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < num_elements){
		C[i] = A[i] + B[i];
	}


}


void sumarVectores(int* A, int* B, int* C, int num_elements){
	//Posicion del thread
	//int i = blockIdx.x * blockDim.x + threadIdx.x;


	for(int i=0; i<num_elements; i++){
		C[i] = A[i] + B[i];
	}
}


int main(){

	int num_elements = 100000;

	//Reservar espacio en memoria HOST


	int * h_A = (int*)malloc(num_elements * sizeof(int));
	int * h_B = (int*)malloc(num_elements * sizeof(int));
	int * h_C = (int*)malloc(num_elements * sizeof(int));



	//Inicializar elementos de los vectores
	for(int i=0; i<num_elements; i++){
		h_A[i] = 1;
		h_B[i] = i;
	}

	hipError_t err;

	int size = num_elements * sizeof(int);

	int * d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	int * d_B = NULL;
	err = hipMalloc((void **)&d_B, size);

	int * d_C = NULL;
	err = hipMalloc((void **)&d_C, size);

	//Copiamos a GPU DEVICE
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

	int HilosPorBloque = 512;
	int BloquesPorGrid = (num_elements + HilosPorBloque -1) / HilosPorBloque;


	//Lanzamos el kernel y medimos tiempos
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	vAdd<<<BloquesPorGrid, HilosPorBloque>>>(d_A, d_B, d_C, num_elements);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float tiempo_reserva_host;
	hipEventElapsedTime(&tiempo_reserva_host, start, stop);


	printf("Tiempo de suma vectores DEVICE: %f\n", tiempo_reserva_host);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	//Copiamos a CPU el vector C
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);



	//Realizamos la suma en la CPU
	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);

	hipEventRecord(start1, 0);

	sumarVectores(h_A, h_B, h_C, num_elements);

	hipEventRecord(stop1,0);
	hipEventSynchronize(stop1);
	float tiempo_reserva_host1;
	hipEventElapsedTime(&tiempo_reserva_host1, start1, stop1);


	printf("Tiempo de suma vectores HOST: %f\n", tiempo_reserva_host1);

	hipEventDestroy(start1);
	hipEventDestroy(stop1);

	/*for(int i=0; i<num_elements; i++){
		printf("%i", h_C[i]);
		printf("\n");
	}*/

}







